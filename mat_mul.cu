
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <iostream>
#include <cstdlib>
#include <ctime>

__device__ float4 MakeFloat4(float val)
{
    return make_float4(val, val, val, val);
}

__device__ float4 operator*(float4 a, float4 b)
{
    return make_float4(a.x * b.x, a.y * b.y, a.z * b.z, a.w * b.w);
}

__device__ float4& operator+=(float4 &a, float4 b)
{
    a = make_float4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
    return a;
}

__global__ void MatMulTopLeft(float *mat0, float *mat1, float *dst, int mat0_col, int w, int h, int stride_num)
{
    // 4*4 
    int global_x = blockIdx.x * blockDim.x + threadIdx.x;
    int global_y = blockIdx.y * blockDim.y + threadIdx.y;
    // int global_x = threadIdx.x;
    // int global_y = threadIdx.y;

    int x = global_x << 2;
    int y = global_y << 2;

    if (x + 4 > w || y + 4 > h)
    {
        return;
    }

    float4 v_zero = MakeFloat4(0);
    float4 v_sum_row0 = v_zero;
    float4 v_sum_row1 = v_zero;
    float4 v_sum_row2 = v_zero;
    float4 v_sum_row3 = v_zero;

    int i = 0;
    for (; i + 4 <= mat0_col; i += 4)
    {
        float4 v_mat0_row0 = *(float4 *)(mat0 + (y + 0) * stride_num + i);
        float4 v_mat0_row1 = *(float4 *)(mat0 + (y + 1) * stride_num + i);
        float4 v_mat0_row2 = *(float4 *)(mat0 + (y + 2) * stride_num + i);
        float4 v_mat0_row3 = *(float4 *)(mat0 + (y + 3) * stride_num + i);

        float4 v_mat1_row0  = *(float4 *)(mat1 + (i + 0) * stride_num + x);
        float4 v_mat1_row1  = *(float4 *)(mat1 + (i + 1) * stride_num + x);
        float4 v_mat1_row2  = *(float4 *)(mat1 + (i + 2) * stride_num + x);
        float4 v_mat1_row3  = *(float4 *)(mat1 + (i + 3) * stride_num + x);

        v_sum_row0 += MakeFloat4(v_mat0_row0.x) * v_mat1_row0;
        v_sum_row1 += MakeFloat4(v_mat0_row1.x) * v_mat1_row0;
        v_sum_row2 += MakeFloat4(v_mat0_row2.x) * v_mat1_row0;
        v_sum_row3 += MakeFloat4(v_mat0_row3.x) * v_mat1_row0;

        v_sum_row0 += MakeFloat4(v_mat0_row0.y) * v_mat1_row1;
        v_sum_row1 += MakeFloat4(v_mat0_row1.y) * v_mat1_row1;
        v_sum_row2 += MakeFloat4(v_mat0_row2.y) * v_mat1_row1;
        v_sum_row3 += MakeFloat4(v_mat0_row3.y) * v_mat1_row1;

        v_sum_row0 += MakeFloat4(v_mat0_row0.z) * v_mat1_row2;
        v_sum_row1 += MakeFloat4(v_mat0_row1.z) * v_mat1_row2;
        v_sum_row2 += MakeFloat4(v_mat0_row2.z) * v_mat1_row2;
        v_sum_row3 += MakeFloat4(v_mat0_row3.z) * v_mat1_row2;

        v_sum_row0 += MakeFloat4(v_mat0_row0.w) * v_mat1_row3;
        v_sum_row1 += MakeFloat4(v_mat0_row1.w) * v_mat1_row3;
        v_sum_row2 += MakeFloat4(v_mat0_row2.w) * v_mat1_row3;
        v_sum_row3 += MakeFloat4(v_mat0_row3.w) * v_mat1_row3;
    }

    for (; i < mat0_col; ++i)
    {
        float val_mat0_row0 = *(float *)(mat0 + (y + 0) * stride_num + i);
        float val_mat0_row1 = *(float *)(mat0 + (y + 1) * stride_num + i);
        float val_mat0_row2 = *(float *)(mat0 + (y + 2) * stride_num + i);
        float val_mat0_row3 = *(float *)(mat0 + (y + 3) * stride_num + i);

        float4 v_mat1_row0  = *(float4 *)(mat1 + (i + 0) * stride_num + x);

        v_sum_row0 += MakeFloat4(val_mat0_row0) * v_mat1_row0;
        v_sum_row1 += MakeFloat4(val_mat0_row1) * v_mat1_row0;
        v_sum_row2 += MakeFloat4(val_mat0_row2) * v_mat1_row0;
        v_sum_row3 += MakeFloat4(val_mat0_row3) * v_mat1_row0;
    }


    *(float4 *)(dst + (y + 0) * stride_num + x) = v_sum_row0;
    *(float4 *)(dst + (y + 1) * stride_num + x) = v_sum_row1;
    *(float4 *)(dst + (y + 2) * stride_num + x) = v_sum_row2;
    *(float4 *)(dst + (y + 3) * stride_num + x) = v_sum_row3;

// printf("****x: %2d, y:%2d, mat0_cold: %d, w: %d, h: %d\n", x, y, mat0_col, w, h);
// if (0 == x && 0 == y)
// {
//     printf("val: %f, %f, %f, %f\n", 
//             mat0[0], mat0[1],
//             mat1[0], mat1[1]
//             );

//     printf("dst: %f, %f, %f, %f\n", 
//             dst[0], dst[1],
//             dst[2], dst[3]
//             );
// }
}


// top-right
__global__ void MatMulTopRight(float *mat0, float *mat1, float *dst, int mat0_col, int w, int h, int stride_num, int offset_x)
{
    // rows: 4, cols: 1
    int global_x = blockIdx.x * blockDim.x + threadIdx.x;
    int global_y = blockIdx.y * blockDim.y + threadIdx.y;

    // int global_x = threadIdx.x;
    // int global_y = threadIdx.y;

    int x = global_x + offset_x;
    int y = global_y << 2;

    if (x >= w || y + 4 > h)
    {
        return;
    }

    float val_sum_row0 = 0;
    float val_sum_row1 = 0;
    float val_sum_row2 = 0;
    float val_sum_row3 = 0;

    int i = 0;
    for (; i + 4 <= mat0_col; i += 4)
    {
        float4 v_mat0_row0 = *(float4 *)(mat0 + (y + 0) * stride_num + i);
        float4 v_mat0_row1 = *(float4 *)(mat0 + (y + 1) * stride_num + i);
        float4 v_mat0_row2 = *(float4 *)(mat0 + (y + 2) * stride_num + i);
        float4 v_mat0_row3 = *(float4 *)(mat0 + (y + 3) * stride_num + i);

        float val_mat1_row0  = *(float *)(mat1 + (i + 0) * stride_num + x);
        float val_mat1_row1  = *(float *)(mat1 + (i + 1) * stride_num + x);
        float val_mat1_row2  = *(float *)(mat1 + (i + 2) * stride_num + x);
        float val_mat1_row3  = *(float *)(mat1 + (i + 3) * stride_num + x);

        val_sum_row0 += v_mat0_row0.x * val_mat1_row0;
        val_sum_row1 += v_mat0_row1.x * val_mat1_row0;
        val_sum_row2 += v_mat0_row2.x * val_mat1_row0;
        val_sum_row3 += v_mat0_row3.x * val_mat1_row0;

        val_sum_row0 += v_mat0_row0.y * val_mat1_row1;
        val_sum_row1 += v_mat0_row1.y * val_mat1_row1;
        val_sum_row2 += v_mat0_row2.y * val_mat1_row1;
        val_sum_row3 += v_mat0_row3.y * val_mat1_row1;

        val_sum_row0 += v_mat0_row0.z * val_mat1_row2;
        val_sum_row1 += v_mat0_row1.z * val_mat1_row2;
        val_sum_row2 += v_mat0_row2.z * val_mat1_row2;
        val_sum_row3 += v_mat0_row3.z * val_mat1_row2;

        val_sum_row0 += v_mat0_row0.w * val_mat1_row3;
        val_sum_row1 += v_mat0_row1.w * val_mat1_row3;
        val_sum_row2 += v_mat0_row2.w * val_mat1_row3;
        val_sum_row3 += v_mat0_row3.w * val_mat1_row3;
    }

    for (; i < mat0_col; ++i)
    {
        float val_mat0_row0 = *(float *)(mat0 + (y + 0) * stride_num + i);
        float val_mat0_row1 = *(float *)(mat0 + (y + 1) * stride_num + i);
        float val_mat0_row2 = *(float *)(mat0 + (y + 2) * stride_num + i);
        float val_mat0_row3 = *(float *)(mat0 + (y + 3) * stride_num + i);

        float val_mat1_row0  = *(float *)(mat1 + (i + 0) * stride_num + x);

        val_sum_row0 += val_mat0_row0 * val_mat1_row0;
        val_sum_row1 += val_mat0_row1 * val_mat1_row0;
        val_sum_row2 += val_mat0_row2 * val_mat1_row0;
        val_sum_row3 += val_mat0_row3 * val_mat1_row0;
    }

    *(float *)(dst + (y + 0) * stride_num + x) = val_sum_row0;
    *(float *)(dst + (y + 1) * stride_num + x) = val_sum_row1;
    *(float *)(dst + (y + 2) * stride_num + x) = val_sum_row2;
    *(float *)(dst + (y + 3) * stride_num + x) = val_sum_row3;
}

// down-left
__global__ void MatMulDownLeft(float *mat0, float *mat1, float *dst, int mat0_col, int w, int h, int stride_num, int offset_y)
{
    // rows: 1, cols: 4
    int global_x = blockIdx.x * blockDim.x + threadIdx.x;
    int global_y = blockIdx.y * blockDim.y + threadIdx.y;

    int x = global_x << 2;
    int y = global_y + offset_y;

    if (x + 4 > w || y >= h)
    {
        return;
    }

    float4 v_sum0 = MakeFloat4(0);
    float4 v_sum1 = MakeFloat4(0);
    float4 v_sum2 = MakeFloat4(0);
    float4 v_sum3 = MakeFloat4(0);

    int i = 0;
    for (; i + 4 <= mat0_col; i += 4)
    {
        float4 v_mat0_row0 = *(float4 *)(mat0 + y * stride_num + i);

        float4 v_mat1_row0  = *(float4 *)(mat1 + (i + 0) * stride_num + x);
        float4 v_mat1_row1  = *(float4 *)(mat1 + (i + 1) * stride_num + x);
        float4 v_mat1_row2  = *(float4 *)(mat1 + (i + 2) * stride_num + x);
        float4 v_mat1_row3  = *(float4 *)(mat1 + (i + 3) * stride_num + x);

        v_sum0 += MakeFloat4(v_mat0_row0.x) * v_mat1_row0;
        v_sum1 += MakeFloat4(v_mat0_row0.y) * v_mat1_row1;
        v_sum2 += MakeFloat4(v_mat0_row0.z) * v_mat1_row2;
        v_sum3 += MakeFloat4(v_mat0_row0.w) * v_mat1_row3;
    }

    for (; i < mat0_col; ++i)
    {
        float val_mat0_row0 = *(float *)(mat0 + y * stride_num + i);

        float4 v_mat1_row0  = *(float4 *)(mat1 + i * stride_num + x);

        v_sum0 += MakeFloat4(val_mat0_row0) * v_mat1_row0;
    }

    v_sum0 += v_sum1;
    v_sum2 += v_sum3;

    v_sum0 += v_sum2;

    *(float4 *)(dst + y * stride_num + x) = v_sum0;
}

// down-right
__global__ void MatMulDownRight(float *mat0, float *mat1, float *dst, int mat0_col, int w, int h, int stride_num, int offset_y, int offset_x)
{
    // rows: 1, cols: 1
    // int global_x = threadIdx.x;
    // int global_y = threadIdx.y;
    int global_x = blockIdx.x * blockDim.x + threadIdx.x;
    int global_y = blockIdx.y * blockDim.y + threadIdx.y;

    int x = global_x + offset_x;
    int y = global_y + offset_y;

    if (x >= w || y >= h)
    {
        return;
    }

    float val_sum0 = 0;
    float val_sum1 = 0;
    float val_sum2 = 0;
    float val_sum3 = 0;

    int i = 0;
    for (; i + 4 <= mat0_col; i += 4)
    {
        float4 v_mat0_row0 = *(float4 *)(mat0 + y * stride_num + i);

        float val_mat1_row0  = *(float *)(mat1 + (i + 0) * stride_num + x);
        float val_mat1_row1  = *(float *)(mat1 + (i + 1) * stride_num + x);
        float val_mat1_row2  = *(float *)(mat1 + (i + 2) * stride_num + x);
        float val_mat1_row3  = *(float *)(mat1 + (i + 3) * stride_num + x);

        val_sum0 += v_mat0_row0.x * val_mat1_row0;
        val_sum1 += v_mat0_row0.y * val_mat1_row1;
        val_sum2 += v_mat0_row0.z * val_mat1_row2;
        val_sum3 += v_mat0_row0.w * val_mat1_row3;
    }

    for (; i < mat0_col; ++i)
    {
        float val_mat0_row0 = *(float *)(mat0 + y * stride_num + i);

        float val_mat1_row0  = *(float *)(mat1 + i * stride_num + x);

        val_sum0 += val_mat0_row0 * val_mat1_row0;
    }

    val_sum0 += val_sum1;
    val_sum2 += val_sum3;

    val_sum0 += val_sum2;

    *(float *)(dst + y * stride_num + x) = val_sum0;
}

void MatMulScalar(float *mat0, float *mat1, float *dst, int mat0_col, int w, int h, int stride_num)
{
    if (!mat0 || !mat1)
    {
        std::cout << "nullptr " << std::endl;
        return;
    }

    for (int y = 0; y < h; ++y)
    {
        for (int x = 0; x < w; ++x)
        {
            float sum = 0.f;

            for (int k = 0; k < mat0_col; ++k)
            {
                sum += mat0[y * stride_num + k] * mat1[k * stride_num + x];
            }

            dst[y * stride_num + x] = sum;
        }
    }
}

void CheckMatVal(float *mat0, float *mat1, int w, int h, int stride_num)
{
    for (int y = 0; y < h; ++y)
    {
        for (int x = 0; x < w; ++x)
        {
            if (std::abs(mat0[y * stride_num + x] - mat1[y * stride_num + x]) > 1e-6)
            {
                std::cout << "mat0: " << mat0[y * stride_num + x] << " ,mat1: " << mat1[y * stride_num + x] << " , y: " << y  << " , x: " << x << std::endl;
                return;
            }
        }
    }

    std::cout << "**** Cmp Ok****" << std::endl;
}

int main()
{
    const int size = 24;

    std::vector<float> mat0(size * size);
    std::vector<float> mat1(size * size);

    std::vector<float> dst_c(size * size);
    std::vector<float> dst_host_cu(size * size);

    // int w = size, h = size, mat0_col = size;
    int w = 23, h = w, mat0_col = w, stride_num = size;

    std::srand(std::time(nullptr));

    for (size_t i = 0; i < mat0.size(); ++i)
    {
        // mat0[i] = std::rand() % size;
        // mat1[i] = std::rand() % size;
        mat0[i] = i;
        mat1[i] = i;
    }

    // gold
    MatMulScalar(mat0.data(), mat1.data(), dst_c.data(), mat0_col, w, h, stride_num);

    // cuda
    // Allocate CUDA mem and copy
    float *mat0_cu, *mat1_cu, *dst_cu;
    size_t bytes_size = mat0.size() * sizeof(float);

    hipMalloc((void **)&mat0_cu,   bytes_size);
    hipMalloc((void **)&mat1_cu,   bytes_size);
    hipMalloc((void **)&dst_cu,    bytes_size);

    hipMemcpy(mat0_cu, mat0.data(), bytes_size, hipMemcpyHostToDevice);
    hipMemcpy(mat1_cu, mat1.data(), bytes_size, hipMemcpyHostToDevice);

    int div_x = w >> 2, div_y = h >> 2;
    int offset_x = div_x << 2;
    int offset_y = div_y << 2;

    int left_x   = w - offset_x;
    int left_y   = h - offset_y;

    dim3 block_size(4, 4); 
    dim3 grid_size((div_x + block_size.x - 1) / block_size.x, (div_y + block_size.y - 1) / block_size.y);
    
    printf("*** block size: %d*%d, grid size: %d*%d\n", block_size.x, block_size.y, grid_size.x, grid_size.y);
    printf("*** w: %d, h: %d, stride_num: %d\n", w, h, stride_num);

    // top left
    MatMulTopLeft<<<grid_size, block_size>>>(mat0_cu, mat1_cu, dst_cu, mat0_col, w, h, stride_num);

    // top right
    if (offset_x < w)
    {
        dim3 block_tr_size(4, 4);
        dim3 grid_tr_size((left_x + block_tr_size.x - 1) / block_tr_size.x, (div_y + block_tr_size.y - 1) / block_tr_size.y);
        MatMulTopRight<<<grid_tr_size, block_tr_size>>>(mat0_cu, mat1_cu, dst_cu, mat0_col, w, h, stride_num, offset_x);
    }

    // down left
    if (offset_y < h)
    {
        dim3 block_dl_size(4, 4);
        dim3 grid_dl_size((div_x + block_dl_size.x - 1) / block_dl_size.x, (left_y + block_dl_size.y - 1) / block_dl_size.y);
        MatMulDownLeft<<<grid_dl_size, block_dl_size>>>(mat0_cu, mat1_cu, dst_cu, mat0_col, w, h, stride_num, offset_y);
    }

    // down right
    if (offset_x < w && offset_y < h)
    {
        dim3 block_dr_size(4, 4);
        dim3 grid_dr_size((left_x + block_dr_size.x - 1) / block_dr_size.x, (left_y + block_dr_size.y - 1) / block_dr_size.y);
        MatMulDownRight<<<grid_dr_size, block_dr_size>>>(mat0_cu, mat1_cu, dst_cu, mat0_col, w, h, stride_num, offset_y, offset_x);
    }

    // Sync CUDA
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess)
    {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(dst_host_cu.data(), dst_cu, bytes_size, hipMemcpyDeviceToHost);

    // Compare
    CheckMatVal(dst_c.data(), dst_host_cu.data(), w, h, stride_num);

    return 0;
}
