#include "hip/hip_runtime.h"
#include "reduce.h"

// 参考实现： https://github.com/PaddleJitLab/CUDATutorial/blob/develop/docs/09_optimize_reduce/02_bank_conflict/README.md

namespace cudaup
{
template<int32_t BLOCK_SIZE>
CUDA_GLOBAL void ReduceBankFreeKernel(Mat *mat, Mat *sum)
{
    __shared__ int32_t shared_data[BLOCK_SIZE];

    int32_t t_id  = threadIdx.x;
    int32_t b_id  = blockIdx.x;
    int32_t b_dim = blockDim.x;
    int32_t g_id  = b_dim * b_id + t_id;

    if (g_id >= mat->m_sizes.m_w)
    {
        return;
    }

    shared_data[t_id] = mat->at<int32_t>(0, g_id);

    __syncthreads();

    // Use bank conflict free
    for (int32_t s = (b_dim >> 1); s > 0; (s >>= 1))
    {
        if (t_id < s)
        {
            shared_data[t_id] += shared_data[t_id + s];
        }

        __syncthreads();
    }

    if (t_id == 0)
    {
        sum->at<int32_t>(0, b_id) = shared_data[0];
    }
}

int32_t ReduceBankConflictFree(Mat &mat, int32_t &sum)
{
    if (mat.empty())
    {
        LOG_ERROR("mat is empty");
        return RET_ERR;
    }

    if (mat.m_sizes.m_ch != 1)
    {
        LOG_ERROR("only support S32C1");
        return RET_ERR;
    }

    Mat sum_mat_gpu = mat.clone(MemType::MEM_GPU);

    const int32_t BLOCK_SIZE = 256; // 至少是warp(32)的整数倍

    dim3 block_size(BLOCK_SIZE);
    dim3 global_size(CEIL_DIV(mat.m_sizes.m_w, BLOCK_SIZE));

    ReduceBankFreeKernel<BLOCK_SIZE><<<global_size, block_size>>>(mat.GetMatAllOnCUDAMem(), sum_mat_gpu.GetMatAllOnCUDAMem());

    Mat sum_mat_cpu = sum_mat_gpu.clone(MemType::MEM_CPU);

    sum = 0;
    for (int32_t i = 0; i < global_size.x; ++i)
    {
        sum += sum_mat_cpu.at<int32_t>(0, i);
    }

    return RET_OK;
}
} // namespace cudaup