#include "hip/hip_runtime.h"
#include "reduce.h"

// 参考实现： https://github.com/PaddleJitLab/CUDATutorial/blob/develop/docs/09_optimize_reduce/01_interleaved_addressing/README.md

namespace cudaup
{
template<int32_t BLOCK_SIZE>
CUDA_GLOBAL void ReduceInterLeaveAddrKernel(Mat *mat, Mat *sum)
{
    __shared__ int32_t shared_data[BLOCK_SIZE];

    int32_t t_id  = threadIdx.x;
    int32_t b_id  = blockIdx.x;
    int32_t b_dim = blockDim.x;
    int32_t g_id  = b_dim * b_id + t_id;

    if (g_id >= mat->m_sizes.m_w)
    {
        return;
    }

    shared_data[t_id] = mat->at<int32_t>(0, g_id);

    __syncthreads();

    // Use interleaved addressing
    for (int32_t s = 1; s < BLOCK_SIZE; s *= 2)
    {
        int32_t index = 2 * s * t_id;

        if (index + s < BLOCK_SIZE && (b_dim * b_id + s < mat->m_sizes.m_w))
        {
            shared_data[index] += shared_data[index + s];
        }

        __syncthreads();
    }

    if (t_id == 0)
    {
        sum->at<int32_t>(0, b_id) = shared_data[0];
    }
}

int32_t ReduceInterLeaveAddr(Mat &mat, int32_t &sum)
{
    if (mat.empty())
    {
        LOG_ERROR("mat is empty");
        return RET_ERR;
    }

    if (mat.m_sizes.m_ch != 1)
    {
        LOG_ERROR("only support S32C1");
        return RET_ERR;
    }

    Mat sum_mat_gpu = mat.clone(MemType::MEM_GPU);

    const int32_t BLOCK_SIZE = 256; // 至少是warp(32)的整数倍

    dim3 block_size(BLOCK_SIZE);
    dim3 global_size(CEIL_DIV(mat.m_sizes.m_w, BLOCK_SIZE));

    ReduceInterLeaveAddrKernel<BLOCK_SIZE><<<global_size, block_size>>>(mat.GetMatAllOnCUDAMem(), sum_mat_gpu.GetMatAllOnCUDAMem());

    Mat sum_mat_cpu = sum_mat_gpu.clone(MemType::MEM_CPU);

    sum = 0;
    for (int32_t i = 0; i < global_size.x; ++i)
    {
        sum += sum_mat_cpu.at<int32_t>(0, i);
    }

    return RET_OK;
}
} // namespace cudaup